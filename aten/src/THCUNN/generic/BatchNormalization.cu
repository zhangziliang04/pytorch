
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/BatchNormalization.cu"
#else

#define DeviceTensor3 THCDeviceTensor<real, 3>
#define DeviceTensor1 THCDeviceTensor<real, 1>

template <int Dim>
static THCDeviceTensor<real, Dim> THNN_(devicetensor)(THCState *state, THCTensor *t) {
  if (!t) {
    return THCDeviceTensor<real, Dim>();
  }

  int inDim = THCTensor_nDimensionLegacyAll(state, t);
  if (inDim == Dim) {
    return toDeviceTensor<real, Dim>(state, t);
  }

  // View in which the last dimensions are collapsed or expanded as needed
  THAssert(THCTensor_isContiguous(state, t));
  int size[Dim];
  for (int i = 0; i < Dim || i < inDim; ++i) {
    if (i < Dim && i < inDim) {
      size[i] = THTensor_sizeLegacyNoScalars(t, i);
    } else if (i < Dim) {
      size[i] = 1;
    } else {
      size[Dim - 1] *= THTensor_sizeLegacyNoScalars(t, i);
    }
  }
  return THCDeviceTensor<real, Dim>(t->data<real>(), size);
}

void THNN_(BatchNormalization_updateOutput)(
  THCState *state, THCTensor *input_, THCTensor *output_,
  THCTensor *weight_, THCTensor *bias_, THCTensor *runningMean_,
  THCTensor *runningVar_, THCTensor *saveMean_, THCTensor *saveStd_,
  bool train, double momentum, double eps) {

  THCTensor_(resizeAs)(state, output_, input_);
  if (train) {
    int64_t nInput = THCTensor_(size)(state, input_, 1);
    THCTensor_(resize1d)(state, saveMean_, nInput);
    THCTensor_(resize1d)(state, saveStd_, nInput);
  }
  DeviceTensor3 input = THNN_(devicetensor)<3>(state, input_);
  DeviceTensor3 output = THNN_(devicetensor)<3>(state, output_);
  DeviceTensor1 weight = THNN_(devicetensor)<1>(state, weight_);
  DeviceTensor1 bias = THNN_(devicetensor)<1>(state, bias_);
  DeviceTensor1 runningMean = THNN_(devicetensor)<1>(state, runningMean_);
  DeviceTensor1 runningVar = THNN_(devicetensor)<1>(state, runningVar_);
  DeviceTensor1 saveMean = THNN_(devicetensor)<1>(state, saveMean_);
  DeviceTensor1 saveStd = THNN_(devicetensor)<1>(state, saveStd_);

  cudaStream_t s = THCState_getCurrentStream(state);
  cudaDeviceProp *prop = THCState_getCurrentDeviceProperties(state);

  if (!train) {
    dim3 blocks(input.getSize(1));
    dim3 threads(getNumThreads(input.getSize(2)));
    BatchNormalizationUpdateOutputInference_kernel<real, accreal, DeviceTensor1, DeviceTensor3> <<<blocks, threads, 0, s>>>(
      input, output, runningMean, runningVar, weight, bias, eps);
  } else {
    dim3 blocks(input.getSize(1));
    dim3 threads(getNumThreads(input.getSize(2)));
    BatchNormalizationUpdateOutput_kernel<real, accreal, DeviceTensor1, DeviceTensor3> <<<blocks, threads, 0, s>>>(
      input, output, weight, bias, eps, momentum, runningMean, runningVar,
      saveMean, saveStd);
  }
  THCudaCheck(cudaGetLastError());
}

void THNN_(BatchNormalization_backward)(
  THCState *state, THCTensor *input_, THCTensor *gradOutput_,
  THCTensor *gradInput_, THCTensor *gradWeight_, THCTensor *gradBias_,
  THCTensor *weight_, THCTensor *runningMean_, THCTensor *runningVar_,
  THCTensor *saveMean_, THCTensor *saveStd_, bool train, double scale, double eps) {

  THCUNN_check_shape(state, input_, gradOutput_);
  if (gradInput_) {
    THCTensor_(resizeAs)(state, gradInput_, input_);
  }

  DeviceTensor3 input = THNN_(devicetensor)<3>(state, input_);
  DeviceTensor3 gradOutput = THNN_(devicetensor)<3>(state, gradOutput_);
  DeviceTensor3 gradInput = THNN_(devicetensor)<3>(state, gradInput_);
  DeviceTensor1 gradWeight = THNN_(devicetensor)<1>(state, gradWeight_);
  DeviceTensor1 gradBias = THNN_(devicetensor)<1>(state, gradBias_);
  DeviceTensor1 weight = THNN_(devicetensor)<1>(state, weight_);
  DeviceTensor1 runningMean = THNN_(devicetensor)<1>(state, runningMean_);
  DeviceTensor1 runningVar = THNN_(devicetensor)<1>(state, runningVar_);
  DeviceTensor1 saveMean = THNN_(devicetensor)<1>(state, saveMean_);
  DeviceTensor1 saveStd = THNN_(devicetensor)<1>(state, saveStd_);

  cudaStream_t s = THCState_getCurrentStream(state);

  dim3 blocks(gradOutput.getSize(1));
  dim3 threads(getNumThreads(gradOutput.getSize(2)));
  BatchNormalizationBackward_kernel<real,  accreal,  DeviceTensor1, DeviceTensor3> <<<blocks, threads, 0, s>>>(
    input, gradOutput, gradInput, gradWeight, gradBias, weight, runningMean, runningVar,
    saveMean, saveStd, train, scale, eps);
  THCudaCheck(cudaGetLastError());
}

#undef DeviceTensor3
#undef DeviceTensor1

#endif
